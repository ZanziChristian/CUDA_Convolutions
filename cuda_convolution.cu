#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define FILTER_WIDTH 3
#define FILTER_RADIUS (FILTER_WIDTH / 2)
#define TILE_WIDTH 32
#define w (TILE_WIDTH + FILTER_WIDTH - 1)
#define clamp(x) (min(max((x), 0.0f), 1.0f))

__constant__ float device_filter[FILTER_WIDTH * FILTER_WIDTH];

__global__ void convolution(float *input, float *output,
                            int channels, int width, int height) {
   __shared__ float N_s[w][w];

   for (int k = 0; k < channels; k++) {
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
          destY = dest / w, destX = dest % w,
          srcY = blockIdx.y * TILE_WIDTH + destY - FILTER_RADIUS,
          srcX = blockIdx.x * TILE_WIDTH + destX - FILTER_RADIUS,
          src = (srcY * width + srcX) * channels + k;

      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         N_s[destY][destX] = input[src];
      else
         N_s[destY][destX] = 0.0f;

      dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
      destY = dest / w; destX = dest % w;
      srcY = blockIdx.y * TILE_WIDTH + destY - FILTER_RADIUS;
      srcX = blockIdx.x * TILE_WIDTH + destX - FILTER_RADIUS;
      src = (srcY * width + srcX) * channels + k;

      if (destY < w) {
         if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_s[destY][destX] = input[src];
         else
            N_s[destY][destX] = 0.0f;
      }
      __syncthreads();

      float accum = 0.0f;
      for (int fy = 0; fy < FILTER_WIDTH; fy++)
         for (int fx = 0; fx < FILTER_WIDTH; fx++)
            accum += N_s[threadIdx.y + fy][threadIdx.x + fx] * device_filter[fy * FILTER_WIDTH + fx];

      int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         output[(y * width + x) * channels + k] = clamp(accum);
      __syncthreads();
   }
}

void launch_convolution(float *host_input, float *host_filter, float *host_output, int channels, int width, int height) {
   float *device_input, *device_output;
   size_t image_size = width * height * channels * sizeof(float);
   size_t filter_size = FILTER_WIDTH * FILTER_WIDTH * sizeof(float);

   hipMalloc(&device_input, image_size);
   hipMalloc(&device_output, image_size);

   hipMemcpy(device_input, host_input, image_size, hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(device_filter), host_filter, filter_size);

   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
   dim3 dimGrid((width + TILE_WIDTH - 1) / TILE_WIDTH, (height + TILE_WIDTH - 1) / TILE_WIDTH);

   std::chrono::time_point<std::chrono::system_clock> start, end;
   start = std::chrono::system_clock::now();
   convolution<<<dimGrid, dimBlock>>>(device_input, device_output, channels, width, height);

   hipDeviceSynchronize();
   end = std::chrono::system_clock::now();
   std::chrono::duration<double> milliseconds = end - start;
   std::cout << "Total convolution time: " << milliseconds.count() << " ms" << std::endl;

   hipError_t err = hipGetLastError();
   if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
   }

   hipMemcpy(host_output, device_output, image_size, hipMemcpyDeviceToHost);


   hipFree(device_input);
   hipFree(device_output);
}

int main(int argc, char** argv) {
   if (argc < 3) {
      std::cerr << "Usage: ./convolve image.jpg output.jpg\n";
      return 1;
   }

   int width, height, channels;
   unsigned char *input_image = stbi_load(argv[1], &width, &height, &channels, 0);
   if (!input_image) {
      std::cerr << "Failed to load image\n";
      return 1;
   }

   // Convert input to float
   size_t image_size = width * height * channels;
   float *host_input = new float[image_size];
   float *host_output = new float[image_size];
   for (size_t i = 0; i < image_size; ++i)
      host_input[i] = input_image[i] / 255.0f;

   // Edge Detection filter
   float host_filter[FILTER_WIDTH * FILTER_WIDTH] = {
      -1, -1, -1,
      -1, 8, -1,
      -1, -1, -1
   };

   launch_convolution(host_input, host_filter, host_output, channels, width, height);

   // Convert back to unsigned char
   unsigned char *output_image = new unsigned char[image_size];
   for (size_t i = 0; i < image_size; ++i)
      output_image[i] = static_cast<unsigned char>(clamp(host_output[i]) * 255.0f);

   stbi_write_jpg(argv[2], width, height, channels, output_image, 100);

   stbi_image_free(input_image);
   delete[] host_input;
   delete[] host_output;
   delete[] output_image;

   std::cout << "Image saved to " << argv[2] << std::endl;
   return 0;
}